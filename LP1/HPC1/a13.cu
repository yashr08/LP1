
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>
#include<stdlib.h>

using namespace std;



__global__ void sum(int *a,int *b,int n)
{


	int block=256*blockIdx.x;
	int sum=0;


	for(int i=block;i<min(block+256,n);i++)
	{


	sum=sum+a[i];


	}
	b[blockIdx.x]=sum;

}


int main()
{


cout<<"Enter the no of elements"<<endl;
int n;
cin>>n;


int a[n];


for(int i=0;i<n;i++)

{

  cout<<"Enter elements: ";
  cin>>a[i];




}


int *ad,*bd;

int size=n*sizeof(int);


hipMalloc(&ad,size);
hipMemcpy(ad,a,size,hipMemcpyHostToDevice);


int grids=ceil(n*1.0f/256.0f);

hipMalloc(&bd,grids*sizeof(int));

dim3 grid(grids,1);
dim3 block(1,1);


int p=n;


hipEvent_t start,end;

hipEventCreate(&start);
hipEventCreate(&end);

hipEventRecord(start);



while(n>1)
{

	sum<<<grid,block>>>(ad,bd,n);
	n=ceil(n*1.0f/256.0f);
	hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);

}


hipEventRecord(end);
hipEventSynchronize(end);


float time=0;


hipEventElapsedTime(&time,start,end);


cout<<"The time is"<<time<<endl;




int add[2];

n=p;


hipMemcpy(add,ad,4,hipMemcpyDeviceToHost);


cout<<"The sum is  "<<add[0]<<endl;

float mean=0.0f;

mean=add[0]/(n*1.0f);

cout<<"The mean is   "<<mean<<endl;



}
