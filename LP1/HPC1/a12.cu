
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;



__global__ void mini1(int *a,int *b,int n)
{


	int block=256*blockIdx.x;


	int mini=7888888;

	for(int i=block;i<min(256+block,n);i++)
	{


		if(mini>a[i])
		{

			mini=a[i];

		}





	}
	b[blockIdx.x]=mini;

}




int main()
{


	cout<<"Enter the size of array"<<endl;
	int n;
	cin>>n;
	int a[n];

	hipEvent_t start,end;

	hipEventCreate(&start);
	hipEventCreate(&end);

	for(int i=0;i<n;i++)
	{

		cout<<"Enter elements: ";
    cin>>a[i];
	}


	int *ad,*bd;
	int size=n*sizeof(int);

	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);



	int grids=ceil(n*1.0f/256.0f);
	hipMalloc(&bd,grids*sizeof(int));



	dim3 grid(grids,1);
	dim3 block(1,1);


	hipEventRecord(start);



	while(n>1)

	{

		mini1<<<grids,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);


	}


	hipEventRecord(end);
	hipEventSynchronize(end);


	float time=0;

	hipEventElapsedTime(&time,start,end);


	int ans[2];

	hipMemcpy(ans,ad,4,hipMemcpyDeviceToHost);

	cout<<"The minimum element is"<<ans[0]<<endl;

	cout<<"The time required dor it is";
	cout<<time<<endl;


}
