#include <hip/hip_runtime.h>

#include <iostream>

#define size 4

using namespace std;

__global__ void add(int *x,int *y,int *z){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if(tid<size){
            z[tid] = x[tid] + y[tid];
        }
}

__global__ void multiplyVectorAndMatrix(int *p, int *q, int *r){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid<size){
        for(int i=0;i<size;i++){
            r[tid] += p[(tid*size)+i] * q[i];
        }
    }
}

__global__ void matrixMultiplication(int *g, int *h, int *ii){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if((row<size) && (col<size)){
        for(int i=0;i<size;i++){
            sum += g[(row*size)+i] * h[(i*size)+col];
        }
	    __syncthreads(); 
        ii[(row*size)+col] = sum;
    }
}

int main(){
    //ADDITION OF TWO VECTORS
    int x[size],y[size],z[size];
    for(int i=0;i<size;i++){
        x[i] = rand()%100+1;
        y[i] = rand()%50+1;
        z[i] = 0;
    }
    cout<<"1st Vector: ";
    for(int i=0;i<size;i++){
        cout<<x[i]<<" ";
    }
    cout<<endl<<"2nd Vector: ";
    for(int i=0;i<size;i++){
        cout<<y[i]<<" ";
    }
    cout<<endl;
    int byte_size = size*sizeof(int);
    cout<<"Addition using CPU: ";
    for(int i=0;i<size;i++){
        cout<<x[i]+y[i]<<" ";
    }
    cout<<endl;
    cout<<"Addition using GPU: ";
    int *a,*b,*c;
    hipMalloc(&a,byte_size);
    hipMemcpy(a,x,byte_size,hipMemcpyHostToDevice);
    hipMalloc(&b,byte_size);
    hipMemcpy(b,y,byte_size,hipMemcpyHostToDevice);
    hipMalloc(&c,byte_size);
    hipMemcpy(c,z,byte_size,hipMemcpyHostToDevice);
    add<<<2,size/2>>>(a,b,c);
    hipMemcpy(&z,c,byte_size,hipMemcpyDeviceToHost);
    for(int i=0;i<size;i++){
        cout<<z[i]<<" ";
    }
    cout<<endl;

    //MULTIPLICATION: MATRIX AND VECTOR
    int m[size][size],n[size],o[size];
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            m[i][j] = rand()%10+1;
        }
        n[i] = rand()%10+1;
        o[i] = 0;
    }
    cout<<endl;
    cout<<"Matrix:"<<endl;
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            cout<<m[i][j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl<<"Vector: ";
    for(int i=0;i<size;i++){
        cout<<n[i]<<" ";
    }
    cout<<endl<<endl;
    size_t matrix_size = size*size*sizeof(int);
    size_t vector_size = size*sizeof(int);
    cout<<"Multiplication using CPU: ";
    for(int i=0;i<size;i++){
        o[i] = 0;
        for(int j=0;j<size;j++){
            o[i]+=m[i][j]*n[j];
        }
    }
    for(int i=0;i<size;i++){
        cout<<o[i]<<" ";
        o[i] = 0;
    }
    cout<<endl;
    cout<<"Multiplication using GPU: ";
    int *p,*q,*r;
    hipMalloc(&p,matrix_size);
    hipMemcpy(p,m,matrix_size,hipMemcpyHostToDevice);
    hipMalloc(&q,vector_size);
    hipMemcpy(q,n,vector_size,hipMemcpyHostToDevice);
    hipMalloc(&r,vector_size);
    hipMemcpy(r,o,vector_size,hipMemcpyHostToDevice);
    multiplyVectorAndMatrix<<<2,size/2>>>(p,q,r);
    hipMemcpy(&o,r,vector_size,hipMemcpyDeviceToHost);
    for(int i=0;i<size;i++){
        cout<<o[i]<<" ";
    }
    cout<<endl;

    //Matrix Multiplication
    int d[size][size],e[size][size],f[size][size];
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            d[i][j] = rand()%10+1;
            e[i][j] = rand()%10+1;
        }
    }
    cout<<endl;
    cout<<"Matrix:"<<endl;
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            cout<<d[i][j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
    cout<<"Matrix:"<<endl;
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            cout<<e[i][j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
    cout<<"Multiplication using CPU:"<<endl;
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            f[i][j] = 0;
            for(int k=0;k<size;k++){
                f[i][j] += d[i][k] * e[k][j];
            }
        }
    }
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            cout<<f[i][j]<<" ";
            f[i][j] = 0;
        }
        cout<<endl;
    }
    cout<<endl;
    cout<<"Multiplication using GPU:"<<endl;
    int *g,*h,*ii;
    hipMalloc(&g,matrix_size);
    hipMemcpy(g,d,matrix_size,hipMemcpyHostToDevice);
    hipMalloc(&h,matrix_size);
    hipMemcpy(h,e,matrix_size,hipMemcpyHostToDevice);
    hipMalloc(&ii,matrix_size);
    hipMemcpy(ii,f,matrix_size,hipMemcpyHostToDevice);
	dim3 threadsPerblock(size,size);
	dim3 blocksPerGrid(1,1);

	if(size*size>512)
	{
	threadsPerblock.x = 512;
	threadsPerblock.y=512;
	blocksPerGrid.x = ceil(double(size)/double(threadsPerblock.x));
	blocksPerGrid.y = ceil(double(size)/double(threadsPerblock.y));

	}
    matrixMultiplication<<<blocksPerGrid,threadsPerblock>>>(g,h,ii);
    hipMemcpy(&f,ii,matrix_size,hipMemcpyDeviceToHost);
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            cout<<f[i][j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
}
